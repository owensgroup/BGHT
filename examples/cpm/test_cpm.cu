#include "hip/hip_runtime.h"
/*
 *   Copyright 2021 The Regents of the University of California, Davis
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */

#include <assert.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <limits>
#include <vector>
#include "bcht.hpp"

int main() {
  using K = unsigned;
  using V = int;
  using pair_type = bght::pair<K, V>;

  auto pair0 = pair_type{static_cast<K>(1), static_cast<V>(5)};
  auto pair1 = pair_type{static_cast<K>(2), static_cast<V>(6)};
  auto pair2 = pair_type{static_cast<K>(3), static_cast<V>(7)};
  auto pair3 = pair_type{static_cast<K>(4), static_cast<V>(8)};

  std::vector<pair_type> h_pairs{pair0, pair1, pair2, pair3};
  thrust::device_vector<pair_type> d_pairs(h_pairs);

  auto sentinel_key = std::numeric_limits<K>::max();
  auto sentinel_value = std::numeric_limits<V>::max();
  bght::bcht<K, V> table(64ull, sentinel_key, sentinel_value);

  bool success =
      table.insert(d_pairs.data().get(), d_pairs.data().get() + d_pairs.size());
  assert(success);

  thrust::device_vector<K> d_queries(std::vector<K>{static_cast<K>(1),
                                                    static_cast<K>(3),
                                                    static_cast<K>(5),
                                                    static_cast<K>(2),
                                                    static_cast<K>(4),
                                                    static_cast<K>(12)});

  thrust::device_vector<V> d_results(d_queries.size());

  table.find(d_queries.data().get(),
             d_queries.data().get() + d_queries.size(),
             d_results.begin());

  thrust::host_vector<V> h_results = d_results;
  assert(h_results[0] == static_cast<V>(5));
  assert(h_results[1] == static_cast<V>(7));
  assert(h_results[2] == static_cast<V>(sentinel_value));
  assert(h_results[3] == static_cast<V>(6));
  assert(h_results[4] == static_cast<V>(8));
  assert(h_results[5] == static_cast<V>(sentinel_value));
  std::cout << "Success\n";
}
